
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <iostream>

__global__ void pairwise_hamming(
    const unsigned char* __restrict__ seqs,
    int N, int M,
    float* __restrict__ out
);

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: ./hamming <binfile> <shapefile>\n";
        return 1;
    }

    int N, M;
    {
        FILE* s = fopen(argv[2], "r");
        if (!s) { perror("shape file"); return 1; }
        fscanf(s, "%d %d", &N, &M);
        fclose(s);
    }

    size_t S = (size_t)N * M;
    unsigned char* hseqs = (unsigned char*)malloc(S);
    {
        FILE* f = fopen(argv[1], "rb");
        if (!f) { perror("bin file"); return 1; }
        fread(hseqs, 1, S, f);
        fclose(f);
    }

    unsigned char *dseqs;
    float *dout;
    hipMalloc(&dseqs, S);
    hipMalloc(&dout, (size_t)N * N * sizeof(float));

    hipMemcpy(dseqs, hseqs, S, hipMemcpyHostToDevice);
    hipMemset(dout, 0, (size_t)N * N * sizeof(float));


    dim3 grid(N, N);
    int tpb = 256;
    size_t shmem = 2 * tpb * sizeof(int);

    pairwise_hamming<<<grid, tpb, shmem>>>(dseqs, N, M, dout);
    hipDeviceSynchronize();


    float* D = (float*)malloc((size_t)N * N * sizeof(float));
    hipMemcpy(D, dout, (size_t)N * N * sizeof(float), hipMemcpyDeviceToHost);

    
    FILE* out = fopen("gpu_dist.bin", "wb");
    fwrite(D, sizeof(float), (size_t)N * N, out);
    fclose(out);

    
    free(hseqs);
    free(D);
    hipFree(dseqs);
    hipFree(dout);

    std::cerr << "GPU computation done\n";
    return 0;
}

__global__ void pairwise_hamming(
    const unsigned char* __restrict__ seqs,
    int N, int M,
    float* __restrict__ out
) {
    int i = blockIdx.x;
    int j = blockIdx.y;
    if (i >= N || j >= N || j <= i) return;

    int mism = 0, valid = 0;

    const unsigned char* a = seqs + i * M;
    const unsigned char* b = seqs + j * M;

    for (int k = threadIdx.x; k < M; k += blockDim.x) {
        unsigned char x = a[k], y = b[k];
        if (x == 255 || y == 255) continue;  // skip gaps
        valid++;
        mism += (x != y);
    }

    // reduction in shared memory
    extern __shared__ int smem[];
    int* mismem = smem;
    int* valmem = smem + blockDim.x;

    mismem[threadIdx.x] = mism;
    valmem[threadIdx.x] = valid;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            mismem[threadIdx.x] += mismem[threadIdx.x + s];
            valmem[threadIdx.x] += valmem[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        float pd = valmem[0] ? float(mismem[0]) / float(valmem[0]) : 0.0f;
        out[(size_t)i * N + j] = pd;
        out[(size_t)j * N + i] = pd;
    }
}
